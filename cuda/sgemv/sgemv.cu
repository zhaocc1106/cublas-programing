#include "hip/hip_runtime.h"
// 通过warp分块实现sgemv

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>

#include "cuda_common.h"

#define GET_TIME_US() \
  std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count()
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

template <typename T>
__inline__ __device__ T WarpReduceSum(T val) {
#pragma unroll
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    val += __shfl_down_sync(0xffffffff, val, offset, warpSize);
  }
  return val;
}

// 矩阵A * 向量x = 向量y
// 矩阵A的大小为m * n, 向量x的大小为n, 向量y的大小为m
// 假设n为32的倍数，通过warp分块实现sgemv， 每一个warp处理矩阵一行
// blockDim.x = 32, blockDim.y = 4
__global__ void SgemvN32(float* __restrict__ A, float* __restrict__ x, float* __restrict__ y, int M, int N) {
  auto tx = threadIdx.x;    // 0 ~ 31
  auto ty = threadIdx.y;    // 0 ~ 4
  auto bx = blockIdx.x;     // 0 ~ m / 32
  int lane = tx % warpSize; // 0 ~ 31

  int m = bx * blockDim.y + ty; // 当前处理的行号
  // printf("m: %d, bx: %d, blockDim.y: %d, ty: %d\n", m, bx, blockDim.y, ty);
  if (m >= M) {
    return;
  }

  float sum = 0;
  auto n_warp = (N + warpSize - 1) / warpSize;
  for (int i = 0; i < n_warp; ++i) {
    auto idx = i * warpSize + lane;
    if (idx < N) {
      sum += A[m * N + idx] * x[idx];
    }
  }
  sum = WarpReduceSum(sum);
  if (lane == 0) {
    y[m] = sum;
  }
}

// 矩阵A * 向量x = 向量y
// 矩阵A的大小为m * n, 向量x的大小为n, 向量y的大小为m
// 假设n为128的倍数，通过warp分块实现sgemv， 每一个warp处理矩阵一行，可以通过FLOAT4优化数据加载
// blockDim.x = 32, blockDim.y = 4
__global__ void SgemvN128(float* __restrict__ A, float* __restrict__ x, float* __restrict__ y, int M, int N) {
  auto tx = threadIdx.x;    // 0 ~ 31
  auto ty = threadIdx.y;    // 0 ~ 4
  auto bx = blockIdx.x;     // 0 ~ m / 32
  int lane = tx % warpSize; // 0 ~ 31

  int m = bx * blockDim.y + ty; // 当前处理的行号
  if (m >= M) {
    return;
  }
  float sum = 0;
  auto n_warp = (N + warpSize - 1) / warpSize / 4;
  for (int i = 0; i < n_warp; ++i) {
    auto idx = (i * warpSize + lane) * 4;
    float r[4], r1[4];
    FLOAT4(r[0]) = FLOAT4(A[m * N + idx]);
    FLOAT4(r1[0]) = FLOAT4(x[idx]);
    if (idx < N) {
      sum += r[0] * r1[0] + r[1] * r1[1] + r[2] * r1[2] + r[3] * r1[3];
    }
  }
  sum = WarpReduceSum(sum);
  if (lane == 0) {
    y[m] = sum;
  }
}

template <typename T>
__inline__ __device__ T HalfWarpReduceSum(T val) {
#pragma unroll
  for (int offset = warpSize / 4; offset > 0; offset /= 2) {
    val += __shfl_down_sync(0xffffffff, val, offset, warpSize);
  }
  return val;
}

// 矩阵A * 向量x = 向量y
// 矩阵A的大小为m * n, 向量x的大小为n, 向量y的大小为m
// 假设n为16的倍数，通过warp分块实现sgemv， 每一个warp处理矩阵两行
// blockDim.x = 32, blockDim.y = 4
__global__ void SgemvN16(float* __restrict__ A, float* __restrict__ x, float* __restrict__ y, int M, int N) {
  auto tx = threadIdx.x;     // 0 ~ 31
  auto ty = threadIdx.y;     // 0 ~ 4
  auto bx = blockIdx.x;      // 0 ~ m / 32
  auto lane = tx % warpSize; // 0 ~ 31

  auto m = (bx * blockDim.y + ty) * 2; // 当前处理的行号
  if (m >= M) {
    return;
  }

  float sum = 0;
  const auto half_warp = warpSize / 2;
  auto n_warp = (N + half_warp - 1) / half_warp;
  for (int i = 0; i < n_warp; ++i) {
    if (lane < half_warp) {
      auto idx = i * half_warp + lane;
      if (idx < N) {
        sum += A[m * N + idx] * x[idx];
      }
    } else {
      auto idx = i * half_warp + lane - half_warp;
      if (idx < N) {
        sum += A[(m + 1) * N + idx] * x[idx];
      }
    }
  }
  if (lane < half_warp) {
    sum = HalfWarpReduceSum(sum);
    if (lane == 0) {
      y[m] = sum;
    }
  } else {
    sum = HalfWarpReduceSum(sum);
    if (lane == half_warp) {
      y[m + 1] = sum;
    }
  }
}

void SgemvCpu(const float* A, const float* x, float* y, int M, int N) {
  for (int i = 0; i < M; ++i) {
    y[i] = 0;
    for (int j = 0; j < N; ++j) {
      y[i] += A[i * N + j] * x[j];
    }
  }
}

template <size_t N = 16>
void Test() {
  int M = 1024;
  float* A = new float[M * N];
  float* A_lead_col = new float[M * N]; // 列式存储
  float* x = new float[N];
  float* y = new float[M];
  float* y_cpu = new float[M];
  float* A_gpu = nullptr;
  float* A_gpu_lead_col = nullptr; // 列式存储
  float* x_gpu = nullptr;
  float* y_gpu = nullptr;
  CUDA_CHECK(hipMalloc(&A_gpu, M * N * sizeof(float)));
  CUDA_CHECK(hipMalloc(&A_gpu_lead_col, M * N * sizeof(float)));
  CUDA_CHECK(hipMalloc(&x_gpu, N * sizeof(float)));
  CUDA_CHECK(hipMalloc(&y_gpu, M * sizeof(float)));

  auto sgemv_fn = SgemvN16;
  if (N == 32) {
    sgemv_fn = SgemvN32;
  } else if (N == 128) {
    sgemv_fn = SgemvN128;
  }

  hipblasHandle_t handle;
  CUBLAS_CHECK(hipblasCreate(&handle));
  long long cpu_avg_time = 0;
  long long sgemv_fn_avg_time = 0;
  long long sgemv_cublas_avg_time = 0;
  for (int repeat = 0; repeat < 11; repeat++) {
    for (int i = 0; i < M * N; ++i) {
      A[i] = rand() % 1000 / 1000.0f;
    }
    // 转换为列式存储
    for (int i = 0; i < M; ++i) {
      for (int j = 0; j < N; ++j) {
        A_lead_col[j * M + i] = A[i * N + j];
      }
    }
    for (int i = 0; i < N; ++i) {
      x[i] = rand() % 1000 / 1000.0f;
    }

    auto begin = GET_TIME_US();
    SgemvCpu(A, x, y_cpu, M, N);
    auto end = GET_TIME_US();
    if (repeat > 0) {
      cpu_avg_time += (end - begin);
    }

    dim3 block(32, 4);
    dim3 grid((M + 4 - 1) / 4, 1);

    memset(y, 0, M * sizeof(float));
    begin = GET_TIME_US();
    CUDA_CHECK(hipMemcpy(A_gpu, A, M * N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(x_gpu, x, N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(y_gpu, 0, M * sizeof(float)));
    sgemv_fn<<<grid, block>>>(A_gpu, x_gpu, y_gpu, M, N);
    CUDA_CHECK(hipMemcpy(y, y_gpu, M * sizeof(float), hipMemcpyDeviceToHost));
    end = GET_TIME_US();
    for (int i = 0; i < M; ++i) {
      if (fabs(y[i] - y_cpu[i]) > 1e-3) {
        std::cout << "sgemv_fn error at " << i << " " << y[i] << " " << y_cpu[i] << std::endl;
      }
    }
    if (repeat > 0) {
      sgemv_fn_avg_time += (end - begin);
    }

    memset(y, 0, M * sizeof(float));
    begin = GET_TIME_US();
    CUBLAS_CHECK(hipblasSetMatrix(M, N, sizeof(float), A_lead_col, M, A_gpu_lead_col, M));
    CUBLAS_CHECK(hipblasSetVector(N, sizeof(float), x, 1, x_gpu, 1));
    CUBLAS_CHECK(hipblasSetVector(M, sizeof(float), y, 1, y_gpu, 1));
    float alpha = 1.0f;
    float beta = 0.0f;
    CUBLAS_CHECK(hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, A_gpu_lead_col, M, x_gpu, 1, &beta, y_gpu, 1));
    CUBLAS_CHECK(hipblasGetVector(M, sizeof(float), y_gpu, 1, y, 1));
    end = GET_TIME_US();
    for (int i = 0; i < M; ++i) {
      if (fabs(y[i] - y_cpu[i]) > 1e-3) {
        std::cout << "SgemvCublas error at " << i << " " << y[i] << " " << y_cpu[i] << std::endl;
      }
    }
    if (repeat > 0) {
      sgemv_cublas_avg_time += (end - begin);
    }
  }
  CUBLAS_CHECK(hipblasDestroy(handle));

  std::cout << "cpu avg time: " << float(cpu_avg_time) / 10.0 << " us, "
            << "sgemv_fn avg time: " << float(sgemv_fn_avg_time) / 10.0 << " us, "
            << "sgemv_cublas avg time: " << float(sgemv_cublas_avg_time) / 10.0 << " us" << std::endl;

  CUDA_CHECK(hipFree(A_gpu));
  CUDA_CHECK(hipFree(x_gpu));
  CUDA_CHECK(hipFree(y_gpu));
  delete[] A;
  delete[] x;
  delete[] y;
  delete[] y_cpu;
}

int main() {
  InitDevice(0);
  std::cout << "N = 16" << std::endl;
  Test<16>();
  std::cout << "N = 32" << std::endl;
  Test<32>();
  std::cout << "N = 128" << std::endl;
  Test<128>();
  return 0;
}
