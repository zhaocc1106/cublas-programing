#include "hip/hip_runtime.h"
// 通过warp tile实现reduce sum.

#include <hip/hip_runtime.h>

#include <cassert>
#include <chrono>
#include <iostream>

#include "cuda_common.h"

#define GET_TIME_US() \
  std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count()

#define WARP_SIZE 32
#define BLOCK_DIM_X 1024
#define GRID_DIM_X 1

template <typename T>
__inline__ __device__ T WarpReduceSum(T val) {
#pragma unroll
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    val += __shfl_down_sync(0xffffffff, val, offset, warpSize);
  }
  return val;
}

template <typename T>
__inline__ __device__ T BlockReduceSum(T val, T* shared) {
  const int laneid = threadIdx.x % warpSize;
  const int warpid = threadIdx.x / warpSize;
  val = WarpReduceSum(val);
  __syncthreads();
  if (laneid == 0) {
    shared[warpid] = val;
  }
  __syncthreads();
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[laneid] : T(0);
  if (warpid == 0) {
    val = WarpReduceSum(val);
  }
  return val;
}

template <typename T>
__global__ void ReduceSum(const T* __restrict__ input, T* __restrict__ output, const int N) {
  __shared__ T shared[BLOCK_DIM_X / WARP_SIZE];
  T sum = T(0);
  // printf("blockDim.x: %d, gridDim.x: %d, i: %d, N: %d\n", blockDim.x, gridDim.x, blockIdx.x * blockDim.x +
  // threadIdx.x, N);
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
    sum += input[i];
  }
  sum = BlockReduceSum(sum, shared);
  if (threadIdx.x == 0) {
    output[blockIdx.x] = sum;
  }
}

template <typename T>
void TestReduceSum(const long long N) {
  T* input = (T*)malloc(N * sizeof(T));
  T* output = (T*)malloc(1 * sizeof(T));
  T* d_input;
  T* d_output;
  CHECK(hipMalloc(&d_input, N * sizeof(T)));
  CHECK(hipMalloc(&d_output, N * sizeof(T)));

  for (int i = 0; i < 10; i++) {
    for (int j = 0; j < N; j++) {
      input[j] = rand();
    }
    auto cpu_begin_us = GET_TIME_US();
    T sum = 0;
    for (int j = 0; j < N; j++) {
      sum += input[j];
    }
    auto cpu_end_us = GET_TIME_US();
    std::cout << "cpu sum: " << sum << ", time: " << cpu_end_us - cpu_begin_us << " us" << std::endl;

    auto gpu_begin_us = GET_TIME_US();
    CHECK(hipMemcpy(d_input, input, N * sizeof(T), hipMemcpyHostToDevice));
    ReduceSum<<<GRID_DIM_X, BLOCK_DIM_X>>>(d_input, d_output, N);
    CHECK(hipMemcpy(output, d_output, 1 * sizeof(T), hipMemcpyDeviceToHost));
    auto gpu_end_us = GET_TIME_US();
    std::cout << "gpu sum: " << output[0] << ", time: " << gpu_end_us - gpu_begin_us << " us" << std::endl;

    assert(std::abs(sum - output[0]) < 1e-3);

    std::cout << std::endl;
  }
  CHECK(hipFree(d_input));
  CHECK(hipFree(d_output));
  free(input);
  free(output);
}

int main() {
  InitDevice(0);

  TestReduceSum<float>(1 << 20);
  return 0;
}