#include "hip/hip_runtime.h"
// CUDA实现sigmoid算子

#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>

#include "cuda_common.h"

#define FLOAT4(val) (reinterpret_cast<float4*>(&(val))[0])
#define GET_US() \
  std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count()

__global__ void Sigmoid(float* __restrict__ a, float* __restrict__ b, int N) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) {
    return;
  }
  b[idx] = 1.0f / (1.0f + expf(-a[idx]));
}

__global__ void SigmoidFloat4(float* __restrict__ a, float* __restrict__ b, int N) {
  auto idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  if (idx >= N) {
    return;
  }
  float4 val = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
  val = FLOAT4(a[idx]);
  val.x = 1.0f / (1.0f + expf(-val.x));
  if (idx + 1 < N) {
    val.y = 1.0f / (1.0f + expf(-val.y));
  }
  if (idx + 2 < N) {
    val.z = 1.0f / (1.0f + expf(-val.z));
  }
  if (idx + 3 < N) {
    val.w = 1.0f / (1.0f + expf(-val.w));
  }
  FLOAT4(b[idx]) = val;
}

void SigmoidCpu(float* a, float* b, int N) {
  for (int i = 0; i < N; i++) {
    b[i] = 1.0f / (1.0f + expf(-a[i]));
  }
}

void Test(int N) {
  auto* a = new float[N];
  auto* b = new float[N];
  float* a_d;
  float* b_d;
  float* b_h;
  CUDA_CHECK(hipMalloc(&a_d, N * sizeof(float)));
  CUDA_CHECK(hipMalloc(&b_d, N * sizeof(float)));
  b_h = new float[N];

  int repeat = 10;
  for (int i = 0; i < repeat; i++) {
    std::cout << "N: " << N << std::endl;
    for (int j = 0; j < N; j++) {
      a[j] = rand() / static_cast<float>(RAND_MAX);
    }
    int block_size = 512;
    int grid_size = (N + block_size - 1) / block_size;
    auto start = GET_US();
    SigmoidCpu(a, b, N);
    auto end = GET_US();
    std::cout << "SigmoidCpu time: " << end - start << " us" << std::endl;

    start = GET_US();
    CUDA_CHECK(hipMemcpy(a_d, a, N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(b_d, 0, N * sizeof(float)));
    Sigmoid<<<grid_size, block_size>>>(a_d, b_d, N);
    CUDA_CHECK(hipMemcpy(b_h, b_d, N * sizeof(float), hipMemcpyDeviceToHost));
    end = GET_US();
    std::cout << "Sigmoid time: " << end - start << " us" << std::endl;
    for (int j = 0; j < N; j++) {
      if (fabs(b[j] - b_h[j]) > 1e-5) {
        std::cout << "Error: " << b[j] << " " << b_h[j] << std::endl;
        break;
      }
    }

    start = GET_US();
    CUDA_CHECK(hipMemcpy(a_d, a, N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(b_d, 0, N * sizeof(float)));
    block_size /= 4;
    grid_size = (N + block_size - 1) / block_size;
    SigmoidFloat4<<<grid_size, block_size>>>(a_d, b_d, N);
    CUDA_CHECK(hipMemcpy(b_h, b_d, N * sizeof(float), hipMemcpyDeviceToHost));
    end = GET_US();
    std::cout << "SigmoidFloat4 time: " << end - start << " us" << std::endl;
    for (int j = 0; j < N; j++) {
      if (fabs(b[j] - b_h[j]) > 1e-5) {
        std::cout << "Error: " << b[j] << " " << b_h[j] << std::endl;
        break;
      }
    }

    std::cout << "--------" << std::endl;
  }

  delete b_h;
  CUDA_CHECK(hipFree(a_d));
  CUDA_CHECK(hipFree(b_d));
  delete[] b;
  delete[] a;
}

int main() {
  InitDevice(0);
  Test(1 << 20);
  return 0;
}
